
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

const int N = 100000;
const int BINS = 10;
const int THREADS_PER_BLOCK = 1000;
const int BLOCKS_PER_GRID = 32;

__global__ void histogram_sharedmem(int *values, int *histo) {
  __shared__ int histo_block[BINS];
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int totalThreads = blockDim.x * gridDim.x;

  // initialize shared memory only once per block
  for (int i = threadIdx.x; i < BINS; i += blockDim.x) {
    histo_block[i] = 0;
  }

  // make other threads wait until the initialization completes
  __syncthreads();

  // process values into the block-wide histogram
  for (int i = tid; i < N; i += totalThreads) {
    atomicAdd(&(histo_block[values[i]]), 1);
  }

  // wait for all threads to finish processing their values
  __syncthreads();

  // add the block's partial results to the global result array
  // that was passed in as an argument
  for (int i = threadIdx.x; i < BINS; i += blockDim.x)  {
    atomicAdd(&(histo[i]), histo_block[i]);
  }
}

int main() {
  int values[N], histo[BINS], *dev_values, *dev_histo;

  hipMalloc((void**)&dev_values, N * sizeof(int));
  hipMalloc((void**)&dev_histo, BINS * sizeof(int));
  hipMemset(dev_histo, 0, BINS * sizeof(int));

  //srand(time(NULL));
  memset(histo, 0, BINS * sizeof(int));
  for (int i = 0; i < N; i++) values[i] = rand() % BINS;

  // copy host data to device
  hipMemcpy(dev_values, values, N * sizeof(int), hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // perform computation on device
  histogram_sharedmem<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dev_values, dev_histo);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float duration_ms;
  hipEventElapsedTime(&duration_ms, start, stop);

  // copy device results to host
  hipMemcpy(histo, dev_histo, BINS * sizeof(int), hipMemcpyDeviceToHost);

  // print results
  for (int i = 0; i < BINS; i++) {
    printf("Bin %d: %d\n", i, histo[i]);
  }
  printf("duration: %3.3fms\n", duration_ms);

  return 0;
}
