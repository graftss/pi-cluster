
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 100000;
const int MASK_SIZE = 5;
const int TRIALS = 100000;

const int THREADS_PER_BLOCK = 1000;
const int NUM_BLOCKS = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

__global__ void convolution_kernel(int *in, int *out, int mask_width, int in_width, int* dev_mask) {
  for (int trial = 0; trial < TRIALS; trial++) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int radius = mask_width / 2;

    int result = 0;
    for (int i = 0, j = tid - radius; i < mask_width && j < in_width; i++, j++) {
      if (j >= 0) {
        result += dev_mask[i] * in[j];
      }
    }

    out[tid] = result;
  }
}

int main() {
  float elapsed_ms;

  int in[N], out[N], *dev_in, *dev_out, *mask, *dev_mask;

  for (int i = 0; i < N; i++) in[i] = i;
  memset(out, 0, N * sizeof(int));
  mask = (int*)malloc(MASK_SIZE * sizeof(int));
  for (int i = 0; i < MASK_SIZE; i++) mask[i] = i + 1;

  hipMalloc((void**)&dev_in, N * sizeof(int));
  hipMemcpy(dev_in, in, N * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)&dev_out, N * sizeof(int));
  hipMemcpy(dev_out, out, N * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)&dev_mask, MASK_SIZE * sizeof(int));
  hipMemcpy(dev_mask, mask, MASK_SIZE * sizeof(int), hipMemcpyHostToDevice);
  free(mask);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  convolution_kernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(
    dev_in, dev_out,
    MASK_SIZE, N,
    dev_mask
  );

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_ms, start, stop);

  hipMemcpy(out, dev_out, N * sizeof(int), hipMemcpyDeviceToHost);

  printf("Input array size: %d\n", N);
  printf("Convolution duration: %3.3fms (%d trials)\n", elapsed_ms, TRIALS);

  return 0;
}