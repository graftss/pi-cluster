
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 100000;
const int TRIALS = 100000;
const int MASK_SIZE = 5;
const int THREADS_PER_BLOCK = 1000;

const int NUM_BLOCKS = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
const int TILE_WIDTH = THREADS_PER_BLOCK + 2 * (MASK_SIZE / 2);

__constant__ int dev_mask[MASK_SIZE];

__global__ void convolution_kernel(int *in, int *out, int mask_width, int in_width) {
  __shared__ int tile[TILE_WIDTH];

  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int radius = mask_width / 2;
  const int tile_offset = blockDim.x * blockIdx.x - radius;

  // initialize the shared memory tile
  if (threadIdx.x == 0) {
    for (int i = 0; i < TILE_WIDTH; i++) {
      int in_index = tile_offset + i;
      if (in_index >= 0 && in_index < in_width) {
        tile[i] = in[in_index];
      } else {
        tile[i] = 0;
      }
    }
  }

  // block other threads until the shared memory is initialized
  __syncthreads();

  // apply the convolution filter using the shared memory tile
  for (int trial = 0; trial < TRIALS; trial++) {
    int result = 0;
    for (int i = 0, j = threadIdx.x; i < mask_width; i++, j++) {
      if (j >= 0) {
        result += dev_mask[i] * tile[j];
      }
    }

    out[tid] = result;
  }
}

int main() {
  float elapsed_ms;

  int in[N], out[N], *dev_in, *dev_out, *mask;

  for (int i = 0; i < N; i++) in[i] = i;
  memset(out, 0, N * sizeof(int));
  mask = (int*)malloc(MASK_SIZE * sizeof(int));
  for (int i = 0; i < MASK_SIZE; i++) mask[i] = i + 1;

  hipMalloc((void**)&dev_in, N * sizeof(int));
  hipMemcpy(dev_in, in, N * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)&dev_out, N * sizeof(int));
  hipMemcpy(dev_out, out, N * sizeof(int), hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(dev_mask), mask, MASK_SIZE * sizeof(int));
  free(mask);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  convolution_kernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(
    dev_in, dev_out,
    MASK_SIZE, N
  );

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_ms, start, stop);

  hipMemcpy(out, dev_out, N * sizeof(int), hipMemcpyDeviceToHost);

  printf("Input size: %d\n", N);
  printf("Convolution duration: %3.3fms (%d trials)\n", elapsed_ms, TRIALS);

  return 0;
}
