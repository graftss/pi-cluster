
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 500000;
const int BLOCKS = 32;
const int THREADS_PER_BLOCK = 256;

__global__ void kernel_dot(float* a, float* b, float* blockPartials) {
  // declare memory that's shared among all threads in the block
  __shared__ float threadPartials[THREADS_PER_BLOCK];

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int totalThreads = blockDim.x * gridDim.x;

  // compute this thread's partial
  int i = tid;
  float threadPartial = 0;
  while (i < N) {
    threadPartial += a[i] * b[i];
    i += totalThreads;
  }

  // store this thread's partial in shared memory
  threadPartials[threadIdx.x] = threadPartial;

  // wait until all threads in the block have computed their partial
  __syncthreads();

  // add together the partials from each thread in the block
  if (threadIdx.x == 0) {
    int blockPartial = 0;
    for (int i = 0; i < THREADS_PER_BLOCK; i++) {
      blockPartial += threadPartials[i];
    }
    blockPartials[blockIdx.x] = blockPartial;
  }
}

int main() {
  // allocate host memory
  float *a, *b, *blockPartials;
  a = (float*) malloc(N * sizeof(float));
  b = (float*) malloc(N * sizeof(float));
  blockPartials = (float*) malloc(BLOCKS * sizeof(float));
  for (int i = 0; i < N; i++) {
    a[i] = 7;
    b[i] = 2;
  }

  // allocate device memory
  float *dev_a, *dev_b, *dev_blockPartials;
  hipMalloc((void**) &dev_a, N * sizeof(float));
  hipMalloc((void**) &dev_b, N * sizeof(float));
  hipMalloc((void**) &dev_blockPartials, BLOCKS * sizeof(float));

  // copy host data to device
  hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

  // set up timing events
  hipEvent_t start, stop;
  float elapsed;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // compute dot product on device
  kernel_dot<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_blockPartials);

  // stop timing and compute elapsed time
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);

  // copy block partials back to host and compute their sum on the host
  hipMemcpy(blockPartials, dev_blockPartials,
    BLOCKS * sizeof(float), hipMemcpyDeviceToHost);
  float result = 0;
  for (int i = 0; i < BLOCKS; i++) {
    result += blockPartials[i];
  }
  printf("result = %3.3f, duration = %3.5f ms\n", result, elapsed);
}
