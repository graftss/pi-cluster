
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 500000;
const int BLOCKS = 32;
const int THREADS_PER_BLOCK = 256;

__global__ void kernel_dot(int* a, int* b, int* blockPartials) {
  // declare memory that's shared among all threads in the block
  __shared__ int threadPartials[THREADS_PER_BLOCK];

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int totalThreads = blockDim.x * gridDim.x;

  // compute this thread's partial
  int i = tid;
  int threadPartial = 0;
  while (i < N) {
    threadPartial += a[i] * b[i];
    i += totalThreads;
  }

  // store this thread's partial in shared memory
  threadPartials[threadIdx.x] = threadPartial;

  // wait until all threads in the block have computed their partial
  __syncthreads();

  // add together the partials from each thread in the block
  if (threadIdx.x == 0) {
    int blockPartial = 0;
    for (int i = 0; i < THREADS_PER_BLOCK; i++) {
      blockPartial += threadPartials[i];
    }
    blockPartials[blockIdx.x] = blockPartial;
  }
}

int main() {
  // allocate host memory
  int *a, *b, *blockPartials;
  a = (int*) malloc(N * sizeof(int));
  b = (int*) malloc(N * sizeof(int));
  blockPartials = (int*) malloc(BLOCKS * sizeof(int));
  for (int i = 0; i < N; i++) {
    a[i] = 7;
    b[i] = 2;
  }

  // allocate device memory
  int *dev_a, *dev_b, *dev_blockPartials;
  hipMalloc((void**) &dev_a, N * sizeof(int));
  hipMalloc((void**) &dev_b, N * sizeof(int));
  hipMalloc((void**) &dev_blockPartials, BLOCKS * sizeof(int));

  // copy host data to device
  hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

  // set up timing events
  hipEvent_t start, stop;
  float elapsed;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // compute dot product on device
  kernel_dot<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_blockPartials);

  // stop timing and compute elapsed time
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);

  // copy block partials back to host and compute their sum on the host
  hipMemcpy(blockPartials, dev_blockPartials,
    BLOCKS * sizeof(int), hipMemcpyDeviceToHost);
  int result = 0;
  for (int i = 0; i < BLOCKS; i++) {
    result += blockPartials[i];
  }
  printf("result = %d, duration = %3.5f ms\n", result, elapsed);
}
